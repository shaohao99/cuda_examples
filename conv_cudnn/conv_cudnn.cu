#include <hipDNN.h>
#include <opencv2/opencv.hpp>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }


cv::Mat load_image(const char* image_path) {   // use opencv to load and normalize an image
    cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

// a function to save output image using opencv
void save_image(const char* output_filename,
                float* buffer,
                int height,
                int width) {
  cv::Mat output_image(height, width, CV_32FC3, buffer);
  // Make negative values zero.
  cv::threshold(output_image,
                output_image,
                /*threshold=*/0,
                /*maxval=*/0,
                cv::THRESH_TOZERO);
  cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
  output_image.convertTo(output_image, CV_8UC3);
  cv::imwrite(output_filename, output_image);
}



int main(int argc, char const *argv[]) {

  hipdnnHandle_t my_cudnn;
  //hipdnnCreate(&my_cudnn);
  checkCUDNN(hipdnnCreate(&my_cudnn));

  // use opencv to load and normalize an image
  cv::Mat image = load_image("/projectnb/scv/shaohao/data/cifar/train/1898_horse.png");

  int batch_size = 1;
  int inp_channels =3;
  int out_channels =3;
  int height = image.rows; 
  int width = image.cols;

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,    // define a 4d tensor to hold an input image
                                      /*format=*/HIPDNN_TENSOR_NHWC,  // layout order of the following arguments
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/1,
                                      /*channels=*/3,
                                      /*image_height=*/image.rows,
                                      /*image_width=*/image.cols));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,    // define a 4d tensor to hold an input image
                                      /*format=*/HIPDNN_TENSOR_NHWC,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/1,
                                      /*channels=*/3,
                                      /*image_height=*/image.rows,
                                      /*image_width=*/image.cols));

  hipdnnFilterDescriptor_t kernel_descriptor;    // define a 4d tensor to hold a filter
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*out_channels=*/3,
                                      /*in_channels=*/3,
                                      /*kernel_height=*/3,
                                      /*kernel_width=*/3));


   hipdnnConvolutionDescriptor_t convolution_descriptor;  // define convolution parameters
   checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
   checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/1,
                                           /*pad_width=*/1,
                                           /*vertical_stride=*/1,
                                           /*horizontal_stride=*/1,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/HIPDNN_CROSS_CORRELATION,
                                           /*computeType=*/HIPDNN_DATA_FLOAT));


  hipdnnConvolutionFwdAlgo_t convolution_algorithm;  // define convolutoin algorithm
  checkCUDNN( hipdnnGetConvolutionForwardAlgorithm(my_cudnn,
                                        input_descriptor,
                                        kernel_descriptor,
                                        convolution_descriptor,
                                        output_descriptor,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        /*memoryLimitInBytes=*/0,
                                        &convolution_algorithm));

  size_t workspace_bytes = 0;   // estimate work space size
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(my_cudnn,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   convolution_descriptor,
                                                   output_descriptor,
                                                   convolution_algorithm,
                                                   &workspace_bytes));
  std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;

// allocate memory
  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int inp_image_bytes = batch_size * inp_channels * height * width * sizeof(float);
  int out_image_bytes = batch_size * out_channels * height * width * sizeof(float);

  float* d_input{nullptr};
  hipMalloc(&d_input, inp_image_bytes);
  hipMemcpy(d_input, image.ptr<float>(0), inp_image_bytes, hipMemcpyHostToDevice);  // copy input image from host to GPU

  float* d_output{nullptr};
  hipMalloc(&d_output, out_image_bytes);
  hipMemset(d_output, 0, out_image_bytes);  // set GPU memory for output image

// Initialize filter
  const float kernel_template[3][3] = {
    {1,  1, 1},
    {1, -8, 1},
    {1,  1, 1}
  };

  float h_kernel[3][3][3][3];
  for (int kernel = 0; kernel < 3; ++kernel) {
    for (int channel = 0; channel < 3; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
  }

  float* d_kernel{nullptr};
  hipMalloc(&d_kernel, sizeof(h_kernel));
  hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);  // copy filter from host to GPU

// Compute convolution on GPU
  const float alpha = 1, beta = 0;  // dstValue = alpha[0]*result + beta[0]*priorDstValue
  checkCUDNN(hipdnnConvolutionForward(my_cudnn, 
                                   &alpha,
                                   input_descriptor,
                                   d_input,
                                   kernel_descriptor,
                                   d_kernel,
                                   convolution_descriptor,
                                   convolution_algorithm,
                                   d_workspace,
                                   workspace_bytes,
                                   &beta,
                                   output_descriptor,
                                   d_output));

// copy output image from GPU memory to CPU memory
  float* h_output = new float[out_image_bytes];
  hipMemcpy(h_output, d_output, out_image_bytes, hipMemcpyDeviceToHost);
  save_image("convolution-out.png", h_output, height, width);  // save output image

// Describe the activation
  hipdnnActivationDescriptor_t activation_descriptor;
  checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
  checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                        /*mode=*/HIPDNN_ACTIVATION_SIGMOID,
                                        /*reluNanOpt=*/HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/0));

// Perform the forward pass of the activation
  checkCUDNN(hipdnnActivationForward(my_cudnn,
                                  activation_descriptor,
                                  &alpha,
                                  output_descriptor,
                                  d_output,
                                  &beta,
                                  output_descriptor,
                                  d_output));

// copy output image from GPU memory to CPU memory
  hipMemcpy(h_output, d_output, out_image_bytes, hipMemcpyDeviceToHost);
  save_image("activation-out.png", h_output, height, width);  // save output image

// Release resources
  delete[] h_output;
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);
  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
  hipdnnDestroyActivationDescriptor(activation_descriptor);
  hipdnnDestroy(my_cudnn);

}  // end of main
